#include "hip/hip_runtime.h"
/*
nvcc hello_parallel.cu -o para
*/

#include <stdio.h>

using namespace std;


int dev_x=1024;
int dev_y=1024;
int dev_z=1;

double *num1_d, *num2_d;

int variable_num = 2;
int pixel_num = dev_x*dev_y;

double num1;
double num2;


__global__ void hello(double* num1, double* num2){
    //int i = blockIdx.x;
    //int j = blockIdx.y;
    int idx = blockIdx.x * blockDim.x + blockIdx.y

    if(i%200000==0){
        num1[idx] = num1[idx] + num2[idx];
    }else{
        num1[idx] = num1[idx] - num2[idx];
    }
}

int main() {
    int blocksize;

    blocksize = dev_y;
    dim3 block (1, 1, 1);
    dim3 grid  (dev_x, dev_y, dev_z);

    size_t data_size = sizeof(double)*dev_x*dev_y;
    hipMalloc((void**) &num1_d, data_size);
    hipMalloc((void**) &num2_d, data_size);
    num1 = (double*) malloc(data_size);
    num2 = (double*) malloc(data_size);

    for(int i=0; i<1048576; i++){
        num1[i] = 0;
        num2[i] = 1;
        //num1_h[i] = num1[i];
        //num2_h[i] = num2[i];
    }

    hipMemcpy(num1_d, num1, data_size, hipMemcpyHostToDevice);
    hipMemcpy(num2_d, num2, data_size, hipMemcpyHostToDevice);


    for(int count=0; count<1; count++){
        hello<<< grid, block >>>(num1_d, num2_d);
        hipDeviceSynchronize();
    }

    hipMemcpy(num1, num1_d, data_size, hipMemcpyDeviceToHost);

    for(int i=0; i<1048576; i=i+100000)printf("No.%d value : %g\n", i, num1[i]);

    hipFree(num1_d);
    hipFree(num2_d);
    //free(num1_h);
    //free(num2_h);

    return 0;
}
