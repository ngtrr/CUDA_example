/*
nvcc hello_parallel.cu -o para
*/


#include <hip/hip_runtime.h>
#include <stdio.h>

using namespace std;

int dev_x=1024;
int dev_y=1024;

double *num1_d, *num2_d;

double num1[1048576];
double num2[1048576];


__global__ void hello(double* num1, double* num2){
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    if(i%200000==0){
        num1[i] = num1[i] + num2[i];
    }else{
        num1[i] = num1[i] - num2[i];
    }
}

int main() {
    int blocksize;

    blocksize = dev_y;
    dim3 block (blocksize, 1, 1);
    dim3 grid  ((dev_x*dev_y)/block.x, 1, 1);

    size_t data_size = sizeof(double)*dev_x*dev_y;
    hipMalloc((void**) &num1_d, data_size);
    hipMalloc((void**) &num2_d, data_size);
    //num1_h = (double*) malloc(data_size);
    //num2_h = (double*) malloc(data_size);

    for(int i=0; i<1048576; i++){
        num1[i] = i;
        num2[i] = i;
        //num1_h[i] = num1[i];
        //num2_h[i] = num2[i];
    }

    hipMemcpy(num1_d, num1, data_size, hipMemcpyHostToDevice);
    hipMemcpy(num2_d, num2, data_size, hipMemcpyHostToDevice);

    hello<<< grid, block >>>(num1_d, num2_d);
    hipDeviceSynchronize();

    hipMemcpy(num1, num1_d, data_size, hipMemcpyDeviceToHost);

    for(int i=0; i<1048576; i=i+100000)printf("No.%d value : %g\n", i, num1[i]);

    hipFree(num1_d);
    hipFree(num2_d);
    //free(num1_h);
    //free(num2_h);

    return 0;
}
